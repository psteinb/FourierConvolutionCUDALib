/*
 * standardCUDAfunctions.cu
 *
 *  Created on: Jul 24, 2014
 *      Author: preibisch
 */
#include "book.h"

#include "convolution3Dfft.h"

//==============================================
 int selectDeviceWithHighestComputeCapability() {

  int numDevices = 0;
  HANDLE_ERROR(hipGetDeviceCount(&numDevices));
  int computeCapability = 0;
  int meta = 0;
  int value = -1;

  hipDeviceProp_t dp;
  for (short devIdx = 0; devIdx < numDevices; ++devIdx) {
    hipGetDeviceProperties(&dp,devIdx);
    meta = 10 * dp.major + dp.minor;
    if (meta > computeCapability) {
      computeCapability = meta;
      value = devIdx;
    }
  }

  return value;
}

 int getCUDAcomputeCapabilityMajorVersion(int devCUDA)
{
    hipDeviceProp_t dp;
    hipGetDeviceProperties(&dp, devCUDA);

	return dp.major;
}

 int getCUDAcomputeCapabilityMinorVersion(int devCUDA)
{

  hipDeviceProp_t dp;
  hipGetDeviceProperties(&dp, devCUDA);

  return dp.minor;
}

 int getNumDevicesCUDA()
{
	int count = 0;
	HANDLE_ERROR(hipGetDeviceCount ( &count ));
	return count;
}

 void getNameDeviceCUDA(int devCUDA, char* name)
{
	hipDeviceProp_t prop;
	HANDLE_ERROR( hipGetDeviceProperties(&prop, devCUDA));

	memcpy(name,prop.name,sizeof(char)*256);
}

 long long int getMemDeviceCUDA(int devCUDA)
{
	hipDeviceProp_t prop;
	HANDLE_ERROR( hipGetDeviceProperties(&prop, devCUDA));
	return ((long long int)prop.totalGlobalMem);
}
